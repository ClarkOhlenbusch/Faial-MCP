#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "DS_timer.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <windows.h>
#include <conio.h>

#define ROW 32
#define COL 32

#define NONE -1
#define DEAD 0
#define LIVE 1
#define PLAYER 2

#define UP 119 // W키
#define LEFT 97 // A키
#define DOWN 115 // S키
#define RIGHT 100 // D키


void initfield(int* _gamefield1, int* _gamefield2, int* Player_Location);
void draw(int* _gamefield);
void Menu();

__global__ void game(int* gamefieldOriginal, int* gamefieldBuffer)
{
	int width = blockDim.x;
	int height = gridDim.x;
	int blockID = threadIdx.x;

	int gridID = blockDim.x * blockIdx.x + blockID;

	if (gamefieldOriginal[gridID] == NONE) {
		gamefieldBuffer[gridID] = NONE;
	}
	else {
		int neighbors = 0;
		if (gamefieldOriginal[gridID - width - 1] == LIVE || gamefieldOriginal[gridID - width - 1] == PLAYER) { // upper left.
			neighbors++;
		}
		if (gamefieldOriginal[gridID - width] == LIVE || gamefieldOriginal[gridID - width] == PLAYER) { // upper.
			neighbors++;
		}
		if (gamefieldOriginal[gridID - width + 1] == LIVE || gamefieldOriginal[gridID - width + 1] == PLAYER) { // upper right.
			neighbors++;
		}
		if (gamefieldOriginal[gridID - 1] == LIVE || gamefieldOriginal[gridID - 1] == PLAYER) { // left.
			neighbors++;
		}
		if (gamefieldOriginal[gridID + 1] == LIVE || gamefieldOriginal[gridID + 1] == PLAYER) { // right.
			neighbors++;
		}
		if (gamefieldOriginal[gridID + width - 1] == LIVE || gamefieldOriginal[gridID + width - 1] == PLAYER) { // lower left.
			neighbors++;
		}
		if (gamefieldOriginal[gridID + width] == LIVE || gamefieldOriginal[gridID + width] == PLAYER) { // lower.
			neighbors++;
		}
		if (gamefieldOriginal[gridID + width + 1] == LIVE || gamefieldOriginal[gridID + width + 1] == PLAYER) { // lower right.
			neighbors++;
		}

		if (gamefieldOriginal[gridID] == DEAD) {
			if (neighbors == 3) {
				gamefieldBuffer[gridID] = LIVE;
			}
		}
		else if (gamefieldOriginal[gridID] == LIVE) {
			if (neighbors < 2 || neighbors > 3) {
				gamefieldBuffer[gridID] = DEAD;
			}
		}
	}

}

__global__ void copy(int* gamefieldOriginal, int* gamefieldBuffer) {
	int width = blockDim.x;
	int height = gridDim.x;
	int blockID = threadIdx.x;

	int gridID = blockDim.x * blockIdx.x + blockID;
	gamefieldOriginal[gridID] = gamefieldBuffer[gridID];
}

int main()
{
	srand(time(NULL));

	int width = COL;
	int height = ROW;
	int size = sizeof(int) * width * height;

	int term = 30;
	int count = 0;

	int* Player_Location = new int;
	*Player_Location = COL + 1; // 플레이어의 시작 지점 설정해주는 곳
	int eat = 0; // 잡아먹은 세포 수
	

	int* gamefieldParallelHost;
	int* gamefieldParallelCUDA;
	int* gamefieldBufferCUDA;
	int* gamefieldPlayer; // 플레이어의 위치를 여기다가 바꿔서 바꾼 결과를 디바이스 메모리에 복사할것임

	hipMalloc(&gamefieldParallelCUDA, size);
	hipMalloc(&gamefieldBufferCUDA, size);

	gamefieldParallelHost = new int[width * height];
	gamefieldPlayer = new int[width * height];

	memset(gamefieldParallelHost, 0, size);
	memset(gamefieldPlayer, 0, size);

	initfield(gamefieldParallelHost, gamefieldPlayer, Player_Location);

	dim3 dimBlock(width);
	dim3 dimGrid(height);

	hipMemcpy(gamefieldBufferCUDA, gamefieldParallelHost, size, hipMemcpyHostToDevice);
	hipMemcpy(gamefieldParallelCUDA, gamefieldParallelHost, size, hipMemcpyHostToDevice);

	char key = '\0'; // 키보드 입력 받을 값 초기화

	Menu();

	while (count < term)
	{
		// 게임을 시작하기전에 먼저 키보드 입력을 받아야 count를 세기 시작하며 게임이 시작된다
		// W : 위로 이동 A: 왼쪽으로 이동 S: 아래로 이동 D : 오른쪽으로 이동
		// 벽 밖으로는 나가지 못한다
		// term 시간 동안 잡아먹은 세포 수를 셀 것임

		hipMemcpy(gamefieldPlayer, gamefieldParallelCUDA, size, hipMemcpyDeviceToHost);

		if (_kbhit()) // 키 입력받으면 true 리턴하는 함수
		{
			// 플레이어의 위치를 Player_temp 변수에 복사
			int Player_temp = *Player_Location;

			key = _getch(); // 입력받은 키 값을 key 변수에 저장

			if (key == UP) // W키 입력시 위로 이동
			{
				Player_temp = Player_temp - COL;
			}
			else if (key == LEFT) // A키 입력시 왼쪽 이동
			{
				Player_temp = Player_temp - 1;
			}
			else if (key == DOWN) // S키 입력시 아래쪽 이동
			{
				Player_temp = Player_temp + COL;
			}
			else if (key == RIGHT) // D키 입력시 오른쪽 이동
			{
				Player_temp = Player_temp + 1;
			}
			else
			{
				printf("잘못 입력했습니다!!!!!\n");
			}

			// 이동하고자 하는 곳이 갈 수 없으면 이동이 불가
			if (gamefieldPlayer[Player_temp] == NONE)
				printf("벽이라 못가요!!!!!!\n");
			else
			{
				if (gamefieldPlayer[Player_temp] == LIVE) // 이동할 곳에 세포가 살아있으면 먹음
					eat++;

				gamefieldPlayer[*Player_Location] = DEAD; // 이동하기전에 원래 있던 곳을 DEAD 상태로 바꿔줌
				*Player_Location = Player_temp; 
				gamefieldPlayer[*Player_Location] = PLAYER; // 이동한 곳을 새로운 플레이어의 위치로 설정

				// 바꾼 gamefieldPlayer를 커널 메모리에 복사해줌
				hipMemcpy(gamefieldParallelCUDA, gamefieldPlayer, size, hipMemcpyHostToDevice); 
				hipMemcpy(gamefieldBufferCUDA, gamefieldPlayer, size, hipMemcpyHostToDevice);
			}
		}

		// 1초마다 커널 함수들이 실행됨(1초마다 맵이 그려짐)
		game << <dimGrid, dimBlock >> > (gamefieldParallelCUDA, gamefieldBufferCUDA);
		copy << <dimGrid, dimBlock >> > (gamefieldParallelCUDA, gamefieldBufferCUDA);

		hipDeviceSynchronize();
		hipMemcpy(gamefieldParallelHost, gamefieldParallelCUDA, size, hipMemcpyDeviceToHost);

		draw(gamefieldParallelHost);
		printf("현재 먹은 세포 수 : %d\n", eat);
		printf("%d초 남았습니다\n", term - count);
		Sleep(1000);
		count++;
		system("cls");
	}
	
	printf("총 먹은 개수 : %d\n", eat);

	hipFree(gamefieldParallelCUDA);
	hipFree(gamefieldBufferCUDA);

	delete[] gamefieldPlayer; delete[] gamefieldParallelHost;
	return 0;
}

void initfield(int* _gamefield1, int* _gamefield2 ,int* _Player_Location)
{
	for (int i = 0; i < ROW * COL; i++)
		_gamefield1[i] = rand() % 2;

	for (int i = 0; i < COL; i++)
	{
		_gamefield1[i] = NONE; // 맨 위
		_gamefield1[i + COL * (ROW - 1)] = NONE; // 맨 아래
	}

	for (int i = 0; i < ROW; i++)
	{
		_gamefield1[COL * i] = NONE; // 맨 왼쪽
		_gamefield1[COL * (i + 1) - 1] = NONE; // 맨 오른쪽
	}

	_gamefield1[*_Player_Location] = PLAYER;

	for (int i = 0; i < ROW * COL; i++) {
		_gamefield2[i] = _gamefield1[i];
	}
}

void draw(int* _gamefield)
{
	for (int i = 0; i < ROW; i++)
	{
		for (int j = 0; j < COL; j++)
		{
			printf("[%2d]", _gamefield[i * ROW + j]);
		}
		printf("\n");
	}
}

void Menu()
{
	printf("-----------------------------------------------------------------------\n");
	printf("\t\t\t세포 먹기 게임입니다\n");
	printf("플레이어의 스타트는 맨 왼쪽 위 위치부터 시작합니다(WASD로 이동해보세요)\n");
	printf("\t\t(주의) 벽으로는 가지 못합니다\n");
	printf("제한 시간내에 세포를 얼마나 많이 먹을 수 있는지 도전해보세요\n");
	printf("\t\t세포들은 계속해서 죽거나 생성됩니다\n");
	printf("-----------------------------------------------------------------------\n");

	for (int i = 0; i < 10; i++)
	{
		printf("%d초 후에 시작됩니다\n", 10 - i);
		Sleep(1000);
	}
	system("cls");
}