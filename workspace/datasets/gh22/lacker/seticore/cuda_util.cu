#include <iostream>

#include "cuda_util.h"

using namespace std;

void checkCuda(const string& tag) {
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    cerr << tag << ": cuda error " << err << ": " << hipGetErrorString(err) << endl;
    exit(1);
  }
}


 
