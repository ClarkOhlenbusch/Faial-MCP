#include "hip/hip_runtime.h"
#include <iostream>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <nvToolsExt.h>
#include "cutil_subset.h"

int main(int argc, char* argv[]) {
  bool p2p_works = true;
  int num_devices = 0;
  CUDA_SAFE_CALL(hipGetDeviceCount(&num_devices));

  // GPUs
  int gpuid_0 = 0;
  int gpuid_1 = 1;

  // Memory Copy Size
  uint32_t size = pow(2, 26); // 2^26 = 67MB

  // Allocate Memory
  uint32_t* dev_0;
  hipSetDevice(gpuid_0);
  hipMalloc((void**)&dev_0, size);

  uint32_t* dev_1;
  hipSetDevice(gpuid_1);
  hipMalloc((void**)&dev_1, size);

  //Check for peer access between participating GPUs: 
  int can_access_peer_0_1;
  int can_access_peer_1_0;
  hipDeviceCanAccessPeer(&can_access_peer_0_1, gpuid_0, gpuid_1);
  hipDeviceCanAccessPeer(&can_access_peer_1_0, gpuid_1, gpuid_0);
  printf("hipDeviceCanAccessPeer(%d->%d): %d\n", gpuid_0, gpuid_1, can_access_peer_0_1);
  printf("hipDeviceCanAccessPeer(%d->%d): %d\n", gpuid_1, gpuid_0, can_access_peer_1_0);

  if (can_access_peer_0_1 && can_access_peer_1_0) {
    // Enable P2P Access
    hipSetDevice(gpuid_0);
    hipDeviceEnablePeerAccess(gpuid_1, 0);
    hipSetDevice(gpuid_1);
    hipDeviceEnablePeerAccess(gpuid_0, 0);
  }

  // Init Timing Data
  uint32_t repeat = 10;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Init Stream
  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  // ~~ Start Test ~~
  hipEventRecord(start, stream);

  //Do a P2P memcpy
  for (int i = 0; i < repeat; ++i) {
    hipMemcpyAsync(dev_0, dev_1, size, hipMemcpyDeviceToDevice, stream);
  }

  hipEventRecord(stop, stream);
  hipStreamSynchronize(stream);
  // ~~ End of Test ~~

  // Check Timing & Performance
  float time_ms;
  hipEventElapsedTime(&time_ms, start, stop);
  double time_s = time_ms / 1e3;

  double gb = size * repeat / (double)1e9;
  double bandwidth = gb / time_s;

  printf("Seconds: %f\n", time_s);
  printf("Unidirectional Bandwidth: %f (GB/s)\n", bandwidth);

  if (can_access_peer_0_1 && can_access_peer_1_0) {
    // Shutdown P2P Settings
    hipSetDevice(gpuid_0);
    hipDeviceDisablePeerAccess(gpuid_1);
    hipSetDevice(gpuid_1);
    hipDeviceDisablePeerAccess(gpuid_0);
  }

  // Clean Up
  hipFree(dev_0);
  hipFree(dev_1);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipStreamDestroy(stream);

/*
#pragma omp parallel num_threads(num_devices) 
  int dev_id = omp_get_thread_num();
  CUDA_SAFE_CALL(hipSetDevice(dev_id));
  CUDA_SAFE_CALL(hipFree(0));
  int canAccessPeer = 0;
  for (int i = 0; i < num_devices; i++) {
    if (i == dev_id) continue;
    CUDA_SAFE_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, i));
    if (canAccessPeer) {
      CUDA_SAFE_CALL(hipDeviceEnablePeerAccess(i, 0));
      std::cerr << "P2P access enabled from " << dev_id << " to " << i << std::endl;
    } else {
      std::cerr << "P2P access not available from " << dev_id << " to " << i << std::endl;
    }
  }
*/
  return 0;
}
