#include "hip/hip_runtime.h"
//pass
//--blockDim=[512,512] --gridDim=1
// https://github.com/TianhuaTao/Megatron-DeepSpeed/commit/5478d67ef2048481e651a05053487fba029c3210
// SOURCE: https://github.com/NVIDIA/Megatron-LM/commit/5478d67ef2048481e651a05053487fba029c3210

template<typename T, typename U, typename V> __global__
void cuComputeGradInput(
    const V* __restrict__ dout,
    const T* __restrict__ input,
    const int n1,
    const int n2,
    const U* __restrict__ mean,
    const U* __restrict__ invvar,
    U epsilon,
    const V* gamma,
    T* grad_input,
    int has_gamma)
{
  __requires(n2 == 1024);
  extern __shared__ U buf[];
  for (auto i1=blockIdx.y; i1 < n1; i1 += gridDim.y) {
    U sum_loss1 = U(0);
    U sum_loss2 = U(0);
    const U c_mean = mean[i1];
    const U c_invvar = invvar[i1];
    const T* k_input = input + i1*n2;
    const V* k_dout = dout + i1*n2;
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    if (has_gamma) {
      int l = 4*thrx;
      for (;  l < n2 - 3;  l+=4*numx) {
        for (int k = 0;  k < 4;  ++k) {
          const U c_h = static_cast<U>(k_input[l+k]);
          const U c_loss = static_cast<U>(k_dout[l+k]);
          sum_loss1 += c_loss * gamma[l+k];
          sum_loss2 += c_loss * gamma[l+k] * (c_h - c_mean) * c_invvar;
        }
      }
      for (;  l < n2;  ++l) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        sum_loss1 += c_loss * gamma[l];
        sum_loss2 += c_loss * gamma[l] * (c_h - c_mean) * c_invvar;
      }
    } else {
      int l = 4*thrx;
      for (;  l+3 < n2;  l+=4*numx) {
        for (int k = 0;  k < 4;  ++k) {
          const U c_h = static_cast<U>(k_input[l+k]);
          const U c_loss = static_cast<U>(k_dout[l+k]);
          sum_loss1 += c_loss;
          sum_loss2 += c_loss * (c_h - c_mean) * c_invvar;
        }
      }
      for (;  l < n2;  ++l) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        sum_loss1 += c_loss;
        sum_loss2 += c_loss * (c_h - c_mean) * c_invvar;
      }
    }
    // intra-warp reductions
    for (int mask = blockDim.x/2;  mask > 0;  mask /= 2) {
      sum_loss1 += WARP_SHFL_XOR(sum_loss1, mask);
      sum_loss2 += WARP_SHFL_XOR(sum_loss2, mask);
    }
    // inter-warp reductions
    if (blockDim.y > 1) {
      //U* buf = shared.getPointer(); 
      for (int offset = blockDim.y/2;  offset > 0;  offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int wrt_i = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
          buf[2*wrt_i] = sum_loss1;
          buf[2*wrt_i+1] = sum_loss2;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.y < offset) {
          const int read_i = threadIdx.y * blockDim.x + threadIdx.x;
          sum_loss1 += buf[2*read_i];
          sum_loss2 += buf[2*read_i+1];
        }
        __syncthreads();
      }
      if (threadIdx.y == 0) {
        buf[2*threadIdx.x] = sum_loss1;
        buf[2*threadIdx.x+1] = sum_loss2;
      }
      __syncthreads();
      if (threadIdx.y !=0) {
        sum_loss1 = buf[2*threadIdx.x];
        sum_loss2 = buf[2*threadIdx.x+1];
      } 
    }
    // all threads now have the two sums over l
    U fH = (U)n2;
    U term1 = (U(1) / fH) * c_invvar;
    T* k_grad_input = grad_input + i1*n2;
    if (has_gamma) {
      for (int l = thrx;  l < n2;  l+=numx) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        U f_grad_input = fH * c_loss * gamma[l];
        f_grad_input -= sum_loss1;
        f_grad_input -= (c_h - c_mean) * c_invvar * sum_loss2;
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    } else {
      for (int l = thrx;  l < n2;  l+=numx) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        U f_grad_input = fH * c_loss;
        f_grad_input -= sum_loss1;
        f_grad_input -= (c_h - c_mean) * c_invvar * sum_loss2;
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    }
    // prevent race where buf is written again before reads are done
    __syncthreads(); // <- BUG
  }
}
