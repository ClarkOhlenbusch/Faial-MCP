#include "hip/hip_runtime.h"
//pass
//--gridDim=[16,16,1]      --blockDim=[8,4,4]

#include "common_short.h"

__global__ void CUDAkernelShortIDCT(short *SrcDst, int ImgStride)
{
    __requires(ImgStride == 512);

    __shared__ short block[KERS_BLOCK_HEIGHT * KERS_SMEMBLOCK_STRIDE];

    int    OffsThreadInRow = IMAD(threadIdx.y, BLOCK_SIZE, threadIdx.x);
    int    OffsThreadInCol = IMUL(threadIdx.z, BLOCK_SIZE);
    int OffsThrRowPermuted = (OffsThreadInRow & 0xFFFFFFE0) | ((OffsThreadInRow << 1) | (OffsThreadInRow >> 4) & 0x1) & 0x1F;

    SrcDst += IMAD(IMAD(blockIdx.y, KERS_BLOCK_HEIGHT, OffsThreadInCol), ImgStride, IMAD(blockIdx.x, KERS_BLOCK_WIDTH, OffsThreadInRow * 2));
    short *bl_ptr = block + IMAD(OffsThreadInCol, KERS_SMEMBLOCK_STRIDE, OffsThreadInRow * 2);

    //load data to shared memory (only first half of threads in each row performs data moving (each thread moves 2 shorts)
    if (OffsThreadInRow < KERS_BLOCK_WIDTH_HALF)
    {
#pragma unroll

        for (int i = 0; i < BLOCK_SIZE; i++)
            ((int *)bl_ptr)[i * (KERS_SMEMBLOCK_STRIDE / 2)] = ((int *)SrcDst)[i * (ImgStride / 2)];
    }

    __syncthreads();
    CUDAshortInplaceIDCT(block + OffsThreadInCol * KERS_SMEMBLOCK_STRIDE + OffsThrRowPermuted, KERS_SMEMBLOCK_STRIDE);
    __syncthreads();
    CUDAshortInplaceIDCT((unsigned int *)(block + OffsThreadInRow * KERS_SMEMBLOCK_STRIDE + OffsThreadInCol));
    __syncthreads();

    //store data to global memory (only first half of threads in each row performs data moving (each thread moves 2 shorts)
    if (OffsThreadInRow < KERS_BLOCK_WIDTH_HALF)
    {
#pragma unroll

        for (int i = 0; i < BLOCK_SIZE; i++)
            ((int *)SrcDst)[i * (ImgStride / 2)] = ((int *)bl_ptr)[i * (KERS_SMEMBLOCK_STRIDE / 2)];
    }
}
