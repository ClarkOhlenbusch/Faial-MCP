#include "hip/hip_runtime.h"
//pass
//--gridDim=128              --blockDim=[128,3,1]

#define MUL(a, b) __umul24(a, b)
#define QRNG_DIMENSIONS 3
#define QRNG_RESOLUTION 31
#define INT_SCALE (1.0f / (float)0x80000001U)

static __constant__ unsigned int c_Table[QRNG_DIMENSIONS][QRNG_RESOLUTION];

__global__ void quasirandomGeneratorKernel(
    float *d_Output,
    unsigned int seed,
    unsigned int N
)
{
    __requires(N == 1048576);
    unsigned int *dimBase = &c_Table[threadIdx.y][0];
    unsigned int      tid = MUL(blockDim.x, blockIdx.x) + threadIdx.x;
    unsigned int  threadN = MUL(blockDim.x, gridDim.x);

    for (unsigned int pos = tid;
         __invariant(tid <= pos),
         __invariant(       pos < N + threadN),
         __invariant(pos % threadN == tid),
         __global_invariant(__write_implies(d_Output, (threadIdx.y * N) <= (__write_offset(d_Output)/sizeof(float)))),
         __global_invariant(__write_implies(d_Output, (__write_offset(d_Output)/sizeof(float) - (threadIdx.y * N)) % threadN == tid)),
         __global_invariant(__write_implies(d_Output, (__write_offset(d_Output)/sizeof(float)/N) == threadIdx.y)),
         pos < N; pos += threadN)
    {
        unsigned int result = 0;
        unsigned int data = seed + pos;

        for (int bit = 0; bit < QRNG_RESOLUTION; bit++, data >>= 1)
            if (data & 1)
            {
                result ^= dimBase[bit];
            }

        d_Output[MUL(threadIdx.y, N) + pos] = (float)(result + 1) * INT_SCALE;
    }
}
